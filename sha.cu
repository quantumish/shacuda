#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstddef>
#include <cstring>
#include <cassert>
#include <array>
#include <bit>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <ios>
#include <string>
#include <vector>
#include <sys/types.h>
#include <sys/stat.h>
#include <unistd.h>
#include <stdio.h>
#include <fcntl.h>
#include <fmt/format.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/types.h>
#include <sys/stat.h>

#define rotr(a,b) (((a) >> (b)) | ((a) << (32-(b))))

// Hacky %b for printf from stackoverflow
//https://stackoverflow.com/questions/111928/is-there-a-printf-converter-to-print-in-binary-format
//https://stackoverflow.com/questions/111928/is-there-a-printf-converter-to-print-in-binary-format/25108449#25108449
#define PRINTF_BINARY_SEPARATOR ""
#define PRINTF_BINARY_PATTERN_INT8 "%c%c%c%c%c%c%c%c"
#define PRINTF_BYTE_TO_BINARY_INT8(i)    \
    (((i) & 0x80ll) ? '1' : '0'), \
    (((i) & 0x40ll) ? '1' : '0'), \
    (((i) & 0x20ll) ? '1' : '0'), \
    (((i) & 0x10ll) ? '1' : '0'), \
    (((i) & 0x08ll) ? '1' : '0'), \
    (((i) & 0x04ll) ? '1' : '0'), \
    (((i) & 0x02ll) ? '1' : '0'), \
    (((i) & 0x01ll) ? '1' : '0')

#define PRINTF_BINARY_PATTERN_INT16 \
    PRINTF_BINARY_PATTERN_INT8               PRINTF_BINARY_SEPARATOR              PRINTF_BINARY_PATTERN_INT8
#define PRINTF_BYTE_TO_BINARY_INT16(i) \
    PRINTF_BYTE_TO_BINARY_INT8((i) >> 8),   PRINTF_BYTE_TO_BINARY_INT8(i)
#define PRINTF_BINARY_PATTERN_INT32 \
    PRINTF_BINARY_PATTERN_INT16              PRINTF_BINARY_SEPARATOR              PRINTF_BINARY_PATTERN_INT16
#define PRINTF_BYTE_TO_BINARY_INT32(i) \
    PRINTF_BYTE_TO_BINARY_INT16((i) >> 16), PRINTF_BYTE_TO_BINARY_INT16(i)
#define PRINTF_BINARY_PATTERN_INT64    \
    PRINTF_BINARY_PATTERN_INT32              PRINTF_BINARY_SEPARATOR              PRINTF_BINARY_PATTERN_INT32
#define PRINTF_BYTE_TO_BINARY_INT64(i) \
    PRINTF_BYTE_TO_BINARY_INT32((i) >> 32), PRINTF_BYTE_TO_BINARY_INT32(i)

#define dump_array32(arr, len) for (size_t DUMP32_ITER = 1; DUMP32_ITER < len+1; DUMP32_ITER++) { \
	printf("" PRINTF_BINARY_PATTERN_INT32 " ", PRINTF_BYTE_TO_BINARY_INT32(*(arr+DUMP32_ITER-1))); \
	if (DUMP32_ITER % 2 == 0) printf("\n");				\
    }									\
    printf("\n")

#define dump_array8(arr, len) for (size_t DUMP8_ITER = 1; DUMP8_ITER < len+1; DUMP8_ITER++) { \
	printf("" PRINTF_BINARY_PATTERN_INT8 " ", PRINTF_BYTE_TO_BINARY_INT8(*(arr+DUMP8_ITER-1))); \
	if (DUMP8_ITER % 8 == 0) printf("\n");					\
    }									\
    printf("\n")

uint64_t k[64] = {
    0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
    0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
    0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
    0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
    0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
    0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
    0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
    0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2,
};

struct sha_ctx {
    uint32_t* hash;
    uint64_t len;

    sha_ctx(uint64_t len);
    void compress(uint32_t* w);
    void dump_hash();
};

sha_ctx::sha_ctx(uint64_t length) :len(length) {
    hipMallocManaged((void**)&hash, 8*sizeof(uint32_t));
    hash[0] = 0x6a09e667;
    hash[1] = 0xbb67ae85;
    hash[2] = 0x3c6ef372;
    hash[3] = 0xa54ff53a;
    hash[4] = 0x510e527f;
    hash[5] = 0x9b05688c;
    hash[6] = 0x1f83d9ab;
    hash[7] = 0x5be0cd19;
}

#define bswap32(x) ((x>>24)&0xff) |		\
    ((x<<8)&0xff0000) |				\
    ((x>>8)&0xff00) |				\
    ((x<<24)&0xff000000)			\

__global__ void process(const uint8_t* bytes, uint32_t* w) {
    size_t chunk = ((blockIdx.y*gridDim.x*blockDim.x)+(blockIdx.x*blockDim.x)+threadIdx.x);
    uint32_t* w_adj = w+(chunk*64);
    memcpy(w_adj, bytes+(chunk*64), 64);
    for (int i = 0; i < 16; i++) w_adj[i] = bswap32(w_adj[i]);
    for (int i = 16; i < 64; i++) {
	uint32_t s0 = (rotr(w_adj[i-15], 7) ^ rotr(w_adj[i-15], 18) ^ (w_adj[i-15] >> 3));
	uint32_t s1 = (rotr(w_adj[i-2], 17) ^ rotr(w_adj[i-2], 19)  ^ (w_adj[i-2] >> 10));
	w_adj[i] = w_adj[i-16] + s0 + w_adj[i-7] + s1;
    }
}

void sha_ctx::compress(uint32_t* w) {
    uint32_t a[8] = {0};
    memcpy(a, hash, 8*sizeof(uint32_t));
    for (int i = 0; i < 64; i++) {
	uint32_t s1 = (rotr(a[4], 6) ^ rotr(a[4], 11) ^ rotr(a[4], 25));
	uint32_t ch = (a[4] & a[5]) ^ ((~a[4]) & a[6]);
	uint32_t temp1 = a[7] + s1 + ch + k[i] + w[i];
	uint32_t s0 = (rotr(a[0], 2) ^ rotr(a[0], 13) ^ rotr(a[0], 22));
	uint32_t maj = (a[0] & a[1]) ^ (a[0] & a[2]) ^ (a[1] & a[2]);
	uint32_t temp2 = s0 + maj;
	for (int i = 7; i > 0; i--) a[i] = a[i-1];
	a[4] += temp1;
	a[0] = temp1 + temp2;
    }
    for (int i = 0; i < 8; i++) {
	hash[i] += a[i];
    }
}

void sha_ctx::dump_hash() {
    uint32_t temp_hash[8];
    memcpy(temp_hash, hash, 8*sizeof(uint32_t));
    for (int i = 0; i < 8; i++) temp_hash[i] = bswap32(temp_hash[i]);
    auto u8_ptr = reinterpret_cast<uint8_t*>(temp_hash);
    for (int i = 0; i < 32; i++) {
	std::cout << fmt::format("{:02x}", u8_ptr[i]);
    }
    std::cout << "\n";
}

int main(int argc, char** argv) {
    int fd = open(argv[1], O_RDONLY | O_NONBLOCK);
    posix_fadvise(fd, 0, 0, POSIX_FADV_SEQUENTIAL);
    struct stat stat;
    fstat(fd, &stat);

    sha_ctx sha(stat.st_size);

    uint8_t* buf;
    constexpr size_t BUFFER_SIZE = 8*8*64*64;
    hipMallocManaged(&buf, BUFFER_SIZE);
    uint32_t* w;
    hipMallocManaged(&w, 4096*(64*4));
    size_t bytes_read = read(fd, buf, BUFFER_SIZE);
    do {
	if (bytes_read == (size_t)-1) {
	    printf("Error reading file.");
	    exit(1);
	}
	if (!bytes_read) break;
	else if (bytes_read < BUFFER_SIZE) {
	    size_t buffer_len = 64 * (((bytes_read + 9) / 64) + 1);
	    for (size_t i = bytes_read; i < buffer_len; i++) buf[i] = 0;
	    buf[bytes_read] = 0b10000000;
	    for (int i = 1; i <= 8; i++) buf[buffer_len-i] = sha.len*8 >> (i-1)*8;
	    size_t num_groups = (buffer_len/64)/1024; 
	    size_t group_shift = 1024*64;
	    for (int i = 0; i < (buffer_len/64)/1024; i++) {
		process<<<1, 1024>>>(buf+(i*group_shift), w+(i*group_shift));
	    }
	    process<<<1, (buffer_len/64)%1024>>>(buf+num_groups*group_shift, w+num_groups*group_shift);	    
	    hipDeviceSynchronize();
	    for (int i = 0; i < buffer_len/64; i++) sha.compress(w+(i*64));
	} else {
	    process<<<dim3{8,8,1}, 64>>>(buf, w);
	    hipDeviceSynchronize();
	    for (int i = 0; i < 4096; i++) sha.compress(w+(i*64));
	}	
	hipError_t err = hipGetLastError();
	if (err != hipSuccess) printf("%s\n", hipGetErrorString(err));
    } while ((bytes_read = read(fd, buf, BUFFER_SIZE)));
    for (int i = 0; i < 8; i++) sha.hash[i] = bswap32(sha.hash[i]);
    auto u8_ptr = reinterpret_cast<uint8_t*>(sha.hash);
    for (int i = 0; i < 32; i++) {
	std::cout << fmt::format("{:02x}", u8_ptr[i]);
    }
    std::cout << "  " << argv[1] << "\n";
}
